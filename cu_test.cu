#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <fstream>
#include <iostream>

#include <hipsolver.h>
#include <iterator>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
using namespace std;


void fun1(double * a, double * b,double * c){
    double l = 1.0;
    double k = 0.0;
    hipblasHandle_t cublasH ;
    hipblasCreate (&cublasH);
    hipblasDgemm(cublasH,
                HIPBLAS_OP_N,HIPBLAS_OP_N,
                2,2,2,
                &l,a,2,
                b,2,
                &k,c,2);
                
}


__global__ void fun6(hipDoubleComplex*A){
    int n = threadIdx.x;
    A[n*32+n] = make_hipDoubleComplex(1.0, 0.0);
}

__global__ void fun7(hipComplex*A, hipDoubleComplex *F,int M){
    int m = threadIdx.x;
    int n = threadIdx.y;
    F[n*M+m] = hipComplexFloatToDouble(A[n*M+m]);
}


//V' * (S^-1)'
__global__ void v_mul_s(hipDoubleComplex * V ,double * S,hipDoubleComplex * VS,int M,int N){
    int m = threadIdx.x;
    int n = threadIdx.y;
    if(m < N&& n < N){
        VS[n * N + m] = hipCmul(hipConj(V[m*N+n]),make_hipDoubleComplex(1.0/S[n],0.0));
    }else{
        VS[n * N + m] = make_hipDoubleComplex(0.0,0.0);
    }
}

void inv(hipDoubleComplex *F,hipDoubleComplex *F_inv,int M){

    double *S;
    hipDoubleComplex * U, * V;
    hipMalloc((void**)&S,M*sizeof(double));
    hipMalloc((void**)&U,M*M*sizeof(hipDoubleComplex));
    hipMalloc((void**)&V,M*M*sizeof(hipDoubleComplex));

    int lwork;
    hipDoubleComplex * work;
    double * rwork = nullptr;
    int *devInfo = nullptr;
    hipsolverHandle_t cusolverH;
    hipsolverDnCreate(&cusolverH);

    hipsolverDnCgesvd_bufferSize(cusolverH,M,M,&lwork);
    hipMalloc((void**)&work,lwork*sizeof(hipDoubleComplex));

    signed char jobu = 'A';
    signed char jobvt = 'A';

    hipsolverDnZgesvd(
        cusolverH,jobu,jobvt,
        M,M,F,M,
        S,
        U,M,
        V,M,
        work,lwork,rwork,
        devInfo
    );
    hipDoubleComplex * VS;
    hipMalloc((void**)&VS, M*M*sizeof(hipDoubleComplex));
    v_mul_s<<<1,dim3(M,M)>>>(V,S,VS,M,M);


    hipDoubleComplex alpha = make_hipDoubleComplex(1.0,0.0);
    hipDoubleComplex beta = make_hipDoubleComplex(0.0,0.0);

    hipblasHandle_t cublasH;
    hipblasCreate(&cublasH);
    hipblasZgemm(
        cublasH,HIPBLAS_OP_N,HIPBLAS_OP_C,
        M,M,M,
        &alpha,
        VS,M,
        U,M,
        &beta, 
        F_inv,M
    );

}


void invv(hipDoubleComplex * F, int M){
    hipsolverHandle_t cusolverH;
    hipsolverDnCreate(&cusolverH);

    hipDoubleComplex *F_inv,*FF;
    hipMalloc((void**)&F_inv,M*M*sizeof(hipDoubleComplex));
    hipMalloc((void**)&FF,M*M*sizeof(hipDoubleComplex));
    hipMemcpy(FF,F,M*M*sizeof(hipDoubleComplex),hipMemcpyDeviceToDevice);


    int lwork;
    hipDoubleComplex * work;
    int *devIpiv = nullptr;
    int *devInfo = nullptr;
    hipsolverDnZgetrf_bufferSize(cusolverH,M,M,F,M,&lwork);
    hipMalloc((void**)&work,lwork*sizeof(hipDoubleComplex));
    hipMalloc((void**)&devIpiv, M * sizeof(int));
    hipsolverDnZgetrf(cusolverH, M, M, F, M, work, devIpiv, devInfo);

    fun6<<<1,M>>>(F_inv);

    hipsolverDnZgetrs(cusolverH, HIPBLAS_OP_N, M, M, F, M, devIpiv, F_inv, M, devInfo);

    hipDoubleComplex test[1024];
    hipMemcpy(test,F_inv,M*M*sizeof(hipDoubleComplex),hipMemcpyDeviceToHost);
    for(int i=0;i<32;i++){
        printf("inv : %lf %lf\n",test[i].x,test[i].y);
    }
    hipblasHandle_t cublasH;
    hipblasCreate(&cublasH);
    hipDoubleComplex alpha = {1.0,0.0};
    hipDoubleComplex beta1 ={0.0,0.0};
    hipDoubleComplex beta2 ={1.0,0.0};
    hipDoubleComplex * E;
    hipMalloc((void**)&E,M*M*sizeof(hipDoubleComplex));
    hipblasZgemm(
        cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,
        M,M,M,&alpha,
        FF,M,
        F_inv,M,
        &beta1,
        E,M
    );
    hipMemcpy(test,E,M*M*sizeof(hipDoubleComplex),hipMemcpyDeviceToHost);
    for(int i=0;i<64;i++){
        printf("E %d: %f %f\n",i,test[i].x,test[i].y);
    }
}




__global__ void fun3(hipComplex * b){
    int k = blockIdx.x;
    int m = threadIdx.x;
    int n = threadIdx.y;
    __shared__ hipComplex s[4];

    if(k < 2)
        s[n*2+m] = make_hipComplex(n*2+m, 0.0);
    else if(k < 4)
        s[n*2+m] = make_hipComplex(2, 1.0);
    //printf("%d %d : %f %f\n",k,m,s[m].x,s[m].y);
    

    atomicAdd(&b[n*2+m].x, s[n*2+m].x);
    atomicAdd(&b[n*2+m].y, s[n*2+m].y);

    // if(k < 5){
    //     b[m] = cuCaddf(b[m], s[m]);
    //     __threadfence();
    // }
    // if(k == 3)
    //     b[m] = cuCaddf(b[m], s[m]);
    // __threadfence();
    
}

__global__ void fun4(int c){
    int m = threadIdx.x;
    hipComplex a = {-3029.961426,10077.462891};
    hipComplex b = hipCmulf(a, hipConjf(a));
    printf("%f %f\n",b.x,b.y);

}

__global__ void norm_Inf(hipDoubleComplex * A,int M, int N,int use_norm,double * res ){
    int m = threadIdx.x; 
    int n = threadIdx.y;

    int len = M*N;
    int idx = m*use_norm+n;
    extern __shared__ double A_s[];
    if(idx < len){
        //printf("%d  = %f %f\n",idx,A_s[idx]);

        A_s[idx] = hipCabs(A[idx]);
        printf("%d  = %f\n",idx,A_s[idx]);
    }
    else{
        A_s[idx] = 0.0f;
    }
    __syncthreads();

    for (int i = M*use_norm/2; i > 0; i >>= 1)
    {
        if(idx < i)
            A_s[idx] = max(A_s[idx],A_s[idx+i]);
        /* code */
        __syncthreads();
    }
    __syncthreads();

    res[0] = A_s[0];
        
}

int main() {

    float *f ;
    float f_h = 0.1;
    int use = 4;

    
    std::vector<hipDoubleComplex> A_h = {
        {1,0},{2, -3},{3,-5},
        {2, 1},{5,2},{3,1}};
    float e_u[ 4 ] = { 1,2,3,4 };
    float e_d[ 4 ] = { 2,2,2,2 };
    std::vector<hipComplex> B_h = {
        {1,0},{2,-3},{3,-5},
        {2,3}};
    int ul_use = 4;

    hipDoubleComplex * a_d, * b_d, * c_d, * B_d;
    hipMalloc((void**)&a_d,6*sizeof(hipDoubleComplex));
    hipMalloc((void**)&c_d,6*sizeof(hipDoubleComplex));
    hipMemcpy(a_d, A_h.data(), 6* sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

    float* e_u_d, * e_d_d;
    hipMalloc((void**)&e_d_d, 4 * sizeof(float));
    hipMalloc((void**)&e_u_d, 4 * sizeof(float));

    hipMemcpy(e_u_d, e_u, ul_use * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(e_d_d, e_d, ul_use * sizeof(float), hipMemcpyHostToDevice);

    double * inf_norm;
    hipMalloc((void**)&inf_norm,sizeof(double));



    norm_Inf<<<1,dim3(2,4),2*4>>>(a_d,2,3,4,inf_norm);
    double cv;
    hipMemcpy(&cv,inf_norm,sizeof(double),hipMemcpyDeviceToHost);
    printf("%f\n",cv);
    //fun4<<<1,1>>>(1);



    // float* sigma_u, * sigma_d;
    // cudaMalloc((void**)&sigma_u, 4 * sizeof(float));
    // cudaMalloc((void**)&sigma_d, ul_use * sizeof(float));
    
    // w_update<<<1, use*2 >>>(sigma_u,sigma_d,e_u_d,e_d_d,use);

    // float w_u[4];
    // float w_d[4];
    // cudaMemcpy(w_u, sigma_u, 4*sizeof(float), cudaMemcpyDeviceToHost);
    // cudaMemcpy(w_d, sigma_d, 4*sizeof(float), cudaMemcpyDeviceToHost);
    // for(int i=0;i<4;i++){
    //     printf("%f : %f %f\n",e_u[i],w_u[i],w_d[i]);
    // }

    // int M = 32;
    // cuDoubleComplex * a_d, * b_d, * c_d, * B_d;
    // cudaMalloc((void**)&a_d,M*M*sizeof(cuDoubleComplex));
    // cudaMalloc((void**)&c_d,M*M*sizeof(cuDoubleComplex));
    // cudaMalloc((void**)&B_d,M*M*sizeof(cuDoubleComplex));
    // cudaMalloc((void**)&b_d, M*M*sizeof(cuDoubleComplex));

    // std::ifstream file("x.txt");

    // cuDoubleComplex A[1024];

    // for(int i=0;i<M;i++){
    //     for(int j=0;j<M;j++){
    //         double x,y;
    //         char c1,c2;
    //         file >> x >> c1 >>y>>c2;
    //         A[j*M+i] = make_cuDoubleComplex(x, y);
    //     }
    // }
    // file.close();

    // for (int i=0;i<65;i++) {
    //     cout<<"F:"<<A[i].x<<" "<<A[i].y<<endl;
    // }
    // cudaMemcpy(a_d,A,M*M*sizeof(cuDoubleComplex),cudaMemcpyHostToDevice);
    // invv(a_d,  M);

    // cuDoubleComplex test[1024];
    // cudaMemcpy(test,c_d,M*M*sizeof(cuDoubleComplex),cudaMemcpyDeviceToHost);
    // for(int i=0;i<32;i++){
    //     printf("inv : %lf %lf\n",test[i].x,test[i].y);
    // }



    // int x ;
    // cudaDeviceProp porp;
    // cudaGetDeviceProperties(&porp, 0);
    // cudaDeviceGetAttribute(&x, cudaDevAttrMaxGridDimX, 0);
    // std::cout << "Device " << 1 << ": " << porp.name << std::endl;
    // std::cout << "Number of cores: " << porp.maxThreadsPerBlock << std::endl;
    // std::cout << "Number of cores: " << x<< std::endl;

    //统计时间
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);  // 记录开始时间

    //fun3<<<4,dim3(2,2)>>>( B_d);


    hipDeviceSynchronize();

    hipEventRecord(stop, 0);   // 记录结束时间
    hipEventSynchronize(stop); // 等待事件完成

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop); // 计算时间差

    printf("run time: %0.4fms\n", milliseconds);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    // cuComplex B[4];
    // cudaMemcpy(B, B_d, 4*sizeof(cuComplex), cudaMemcpyDeviceToHost);
    // for (int i=0; i<4;i++) {
    //     printf("%f %f\n",B[i].x,B[i].y);
    // }



    return 0;
}