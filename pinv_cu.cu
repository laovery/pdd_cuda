#include<iostream>
#include"hip/hip_runtime.h"
#include<hipblas.h>
#include<stdlib.h>
#include<time.h>
 
 
//矩阵的阶数
#define N 3
//有两个矩阵
#define NUM 2
 
int main()
{
	//开辟一个二维的数组空间
	float **matHost = new float*[NUM];
	for(int i=0;i<NUM;i++)
		matHost[i] = new float[N*N];
	
	//matHost[0] = {-0.997497,0.617481,-0.299417,0.127171,0.170019,
	//0.791925,-0.613392,-0.0402539,0.64568};
	matHost[0][0] = -0.997497;
	matHost[0][1] = 0.617481;
	matHost[0][2] = -0.299417;
	matHost[0][3] = 0.127171;
	matHost[0][4] = 0.170019;
	matHost[0][5] = 0.791925;
	matHost[0][6] = -0.613392;
	matHost[0][7] = -0.0402539;
	matHost[0][8] = 0.64568;
 
 
	//随机初始化矩阵，所有矩阵被初始化成一样的
	for(int j=1;j<NUM;j++)
	{
		for(int i=0;i<N*N;i++)
		{
			matHost[j][i] = matHost[0][i];
		}
	}	
 
	//指针在host端，内容却在device端
	float **srchd = new float*[NUM];
	
	for(int i=0;i<NUM;i++)
	{
		hipMalloc((void**)&srchd[i],sizeof(float)*N*N);
		hipMemcpy(srchd[i],matHost[i],sizeof(float)*N*N,hipMemcpyHostToDevice);
	}
 
	float **srcDptr;
	hipMalloc((void**)&srcDptr,sizeof(float*)*NUM);
	hipMemcpy(srcDptr,srchd,sizeof(float*)*NUM,hipMemcpyHostToDevice);
 
 
	//用来记录LU分解是否成功，0表示分解成功
	int *infoArray;
	hipMalloc((void**)&infoArray,sizeof(int)*NUM);
 
	int *pivotArray;
	hipMalloc((void**)&pivotArray,sizeof(int)*N*NUM);
 
	hipblasHandle_t cublasHandle;
	hipblasCreate(&cublasHandle); 
 
	//LU分解,原地的
	hipblasSgetrfBatched(cublasHandle,N,srcDptr,N,pivotArray,infoArray,NUM);
 
	float **resulthd = new float*[NUM];
	for(int i=0;i<NUM;i++)
		hipMalloc((void**)&resulthd[i],sizeof(float)*N*N);
 
	float **resultDptr;
	hipMalloc((void**)&resultDptr,sizeof(float*)*NUM);
	hipMemcpy(resultDptr,resulthd,sizeof(float*)*NUM,hipMemcpyHostToDevice);
 
	//把LU分解的结果变成逆矩阵
	hipblasSgetriBatched(cublasHandle,N,(const float**)srcDptr,N,pivotArray,resultDptr,N,infoArray,NUM);
 
	float **invresult = new float*[NUM];
	for(int i=0;i<NUM;i++)
	{
		invresult[i] = new float[N*N];
		//注意是resulthd[i]而不是resultDptr[i]，否则会出错
		hipMemcpy(invresult[i],resulthd[i],sizeof(float)*N*N,hipMemcpyDeviceToHost);
	}
		
 
	int *infoArrayHost = new int[NUM];
	hipMemcpy(infoArrayHost,infoArray,sizeof(int)*NUM,hipMemcpyDeviceToHost);
 
	std::cout<<"info array:"<<std::endl;
	for(int i=0;i<NUM;i++)
		std::cout<<infoArrayHost[i]<<"  ";
	std::cout<<std::endl;
 
	hipblasDestroy(cublasHandle);
 
	std::cout<<"LU decomposition result:"<<std::endl;
	for(int i=0;i<N*N;i++)
	{	
		if(i%N == 0)
			std::cout<<std::endl;
 
		std::cout<<invresult[0][i]<<"  ";	
	}
	std::cout<<std::endl;
 
	//释放空间
	for(int i=0;i<NUM;i++)
	{
		hipFree(srchd[i]);
		delete []matHost[i];
		matHost[i] = NULL;
		hipFree(resulthd[i]);
		delete []invresult[i];
		invresult[i] = NULL;
	}
 
	delete []matHost;
	matHost = NULL;
	delete []resulthd;
	resulthd = NULL;
	delete []invresult;
	invresult = NULL;
 
	delete []infoArrayHost;
	infoArrayHost = NULL;
 
	delete []srchd;
	srchd = NULL;
	
	hipFree(infoArray);
	hipFree(pivotArray);
	hipFree(srcDptr);
	hipFree(resultDptr);
 
	return 0;
 
}