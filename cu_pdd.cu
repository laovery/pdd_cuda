#include "hip/hip_runtime.h"
#include <complex>
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hipsolver.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <fstream>

#include "cu_pdd.h"


__device__ hipDoubleComplex global_v[32] = {0,0.0};



__global__ void w_update(
    double * w_u, 
    double * w_d, 
    double * e_u, 
    double * e_d, 
    int use) {
    int idx = threadIdx.x;
    if (idx < use) {
        w_u[ idx ] = 1.0 / (log(2.0) * e_u[ idx ]);
    }
    else if (idx < 2 * use) {
        w_d[ idx - use ] = 1.0 / (log(2.0)  * e_d[ idx - use ]);
    }
}

__global__ void p_update(
    double* p_u, 
    double* w_u,
    double* w_d,
    hipDoubleComplex* v_u,
    hipDoubleComplex* v_d,
    hipDoubleComplex* H_u,
    hipDoubleComplex* H1, 
    hipDoubleComplex* H3,
    double I_th,
    int M,
    int use)
    {
    //<<<(4,4);32>>>
    int k = blockIdx.x;
    int s = blockIdx.y;
    int n = threadIdx.x;

    global_v[n] = {0.0,0.0};


    __shared__ hipDoubleComplex H_H[32];
    H_H[n] = hipCmul(hipConj(v_u[s * M + n]) , H_u[k * M + n]);
    //printf("%d %d %d: %f %f\n",k,s,n,v_u[s * M + n].x,v_u[s * M + n].y);
    

    __syncthreads();

    
    for (size_t i = M/2; i > 0; i >>= 1)
    {
        if(n < i){
            H_H[n] = hipCadd(H_H[n] , H_H[n+i]);
        }
        /* code */
        __syncthreads();
    }
    __syncthreads();

    if(n == 0){
        H_H[0] = hipCmul(make_hipDoubleComplex(w_u[s], 0.0), hipCmul(H_H[0], hipConj(H_H[0])));
        
        hipDoubleComplex hh = hipCmul(v_d[s], H1[k*use+s]);
        hh = hipCmul(make_hipDoubleComplex(w_d[s], 0.0), hipCmul(hh, hipConj(hh)));
        hh = hipCadd(hh, H_H[0]);
        global_v[k*use+s] = hh;
    }
    __syncthreads();
    __shared__ double B[32];

    hipDoubleComplex b;
    b = hipCmul(hipConj(v_u[k*M+n]), H_u[k*M+n]);
    B[n] = b.x;
    __syncthreads();

    for (size_t i = M/2; i > 0; i >>= 1)
    {
        if(n < i){
            B[n] = B[n] + B[n + i];
        }
        __syncthreads();
        /* code */
    }
    __syncthreads();


    double bb = B[0]*w_u[k];


    if(s == 0 && n == 0){

        hipDoubleComplex a = {0.0,0.0};
        for(int i=0;i<use;i++){
            a = hipCadd(a, global_v[k*use+i]);
        }

        hipDoubleComplex b = make_hipDoubleComplex(bb, 0.0);

        if(b.x >= 0){
            double p_use = 1.0;
            double b_a = pow(hipCabs(hipCdiv(b , a)), 2);
            double I_h3 = I_th / hipCreal(hipCmul(H3[k], hipConj(H3[k])));

            p_u[k] = min(min(p_use, b_a) , I_h3);
        }
        else
            p_u[k] = 0;

    }
    __syncthreads();

}


//F_update
//按照列存储运算，输出参数为列存储
__global__ void XY_update(
    hipDoubleComplex * X,
    hipDoubleComplex * H,
    hipDoubleComplex * Y, 
    hipDoubleComplex * v_u, 
    hipDoubleComplex * v_d, 
    double * w_u,
    double * w_d,
    hipDoubleComplex * H_d,
    int use){
    
    int k = blockIdx.x;
    int m = threadIdx.x;
    int n = threadIdx.y;
    int M = 32;

    if(k == 0){
        __shared__ hipDoubleComplex w[1024];
        hipDoubleComplex ww;
        for(int i=0;i<use;i++){
            ww = hipCmul(v_u[i*M+m],hipConj(v_u[i*M+n]));
            ww = hipCmul(ww, make_hipDoubleComplex(w_u[i],0.0));
            w[n*M+m] = hipCadd(w[n*M+m], ww);
            __syncthreads();
        }
        X[n*M+m] = w[n*M+m];
        if (m == n) {
            X[n*M+m].y = 0.0;
        }
    }
    else if (k == 1) {
        __shared__ hipDoubleComplex h[1024];
        hipDoubleComplex hh;
        for(int i=0;i<use;i++){
            hh = hipCmul(hipConj(v_d[i]), v_d[i]);
            hh = hipCmul(hipConj(H_d[m*use+i]), hh);
            hh = hipCmul(hh, H_d[n*use+i]);
            h[n*M+m] = hipCadd(h[n*M+m], hipCmul(make_hipDoubleComplex(w_d[i], 0.0), hh));
            __syncthreads();
        }
        H[n*M+m] = h[n*M+m];
    }
    else {
        if (m < 32 && n < use) {
            hipDoubleComplex yy;
            yy = hipCmul(hipConj(H_d[m*use+n]), hipConj(v_d[n]));
            Y[n*M+m] = hipCmul(make_hipDoubleComplex(w_d[n],0.0), yy);
        }
    }
      
}

__global__ void mat_add_I(hipDoubleComplex * x, float p){
    int m = threadIdx.x;
    int n = threadIdx.y;


    int M = 32;
    if(m == n){
        x[m*M+n] = hipCadd(x[m*M+n], make_hipDoubleComplex(p, 0.0));
        x[m*M+n].y = 0.0;
    }
}

// void inv(hipComplex * F, hipComplex * F_inv, int M){
//     int num = 1; 
//     hipblasHandle_t cublasH;
//     hipblasCreate(&cublasH);

//     hipComplex ** A = new hipComplex*[num];
//     A[0] = F;
//     hipComplex ** A_d ;
//     hipMalloc((void**)&A_d, num * sizeof(hipComplex *));
//     hipMemcpy(A_d, A, num * sizeof(hipComplex *), hipMemcpyHostToDevice);

//     int *info;
//     int *pivot;
//     hipMalloc((void**)&info, num * sizeof(int));
//     hipMalloc((void**)&pivot, M * num * sizeof(int));

//     //LU分解
//     hipblasCgetrfBatched(cublasH, M, A_d, M, pivot, info, num);

//     hipComplex ** res = new hipComplex *[num];
//     res[0] = F_inv;
//     hipComplex ** res_d;
//     hipMalloc((void**)&res_d, num * sizeof(hipComplex *));
//     hipMemcpy(res_d, res, num*sizeof(hipComplex *), hipMemcpyHostToDevice);

    
//     //求逆
//     hipblasCgetriBatched(cublasH, M, A_d, M, pivot, res_d, M, info, num );

// }

__global__ void fun6(hipDoubleComplex*A,int M){
    int n = threadIdx.x;
    A[n*M+n] = make_hipDoubleComplex(1.0, 0.0);
}



void inv(hipDoubleComplex * F, hipDoubleComplex * F_inv, int M){
    hipsolverHandle_t cusolverH;
    hipsolverDnCreate(&cusolverH);


    int lwork;
    hipDoubleComplex * work;
    int *devIpiv = nullptr;
    int *devInfo = nullptr;
    hipsolverDnZgetrf_bufferSize(cusolverH,M,M,F,M,&lwork);
    
    hipMalloc((void**)&work,lwork*sizeof(hipDoubleComplex));
    hipMalloc((void**)&devIpiv, M * sizeof(int));
    hipsolverDnZgetrf(cusolverH, M, M, F, M, work, devIpiv, devInfo);

    fun6<<<1,M>>>(F_inv,M);

    hipsolverDnZgetrs(cusolverH, HIPBLAS_OP_N, M, M, F, M, devIpiv, F_inv, M, devInfo);

}





// void inv(hipComplex *F,hipComplex *F_inv,int M){

//     float *S;
//     hipComplex * U, * V;
//     hipMalloc((void**)&S,M*sizeof(float));
//     hipMalloc((void**)&U,M*M*sizeof(hipComplex));
//     hipMalloc((void**)&V,M*M*sizeof(hipComplex));

//     int lwork;
//     hipComplex * work;
//     float * rwork = nullptr;
//     int *devInfo = nullptr;
//     hipsolverHandle_t cusolverH;
//     hipsolverDnCreate(&cusolverH);

//     hipsolverDnCgesvd_bufferSize(cusolverH,M,M,&lwork);
//     hipMalloc((void**)&work,lwork*sizeof(hipComplex));

//     signed char jobu = 'A';
//     signed char jobvt = 'A';

//     hipsolverDnCgesvd(
//         cusolverH,jobu,jobvt,
//         M,M,F,M,
//         S,
//         U,M,
//         V,M,
//         work,lwork,rwork,
//         devInfo
//     );
//     hipComplex * VS;
//     hipMalloc((void**)&VS, M*M*sizeof(hipComplex));
//     v_mul_s<<<1,dim3(M,M)>>>(V,S,VS,M,M);


//     hipComplex alpha = make_hipComplex(1.0,0.0);
//     hipComplex beta = make_hipComplex(0.0,0.0);

//     hipblasHandle_t cublasH;
//     hipblasCreate(&cublasH);
//     hipblasCgemm(
//         cublasH,HIPBLAS_OP_N,HIPBLAS_OP_C,
//         M,M,M,
//         &alpha,
//         VS,M,
//         U,M,
//         &beta, 
//         F_inv,M
//     );

// }



hipDoubleComplex * F_update(
    hipDoubleComplex * F, 
    hipDoubleComplex * F_RF,
    hipDoubleComplex * F_BB,
    hipDoubleComplex * v_u, 
    hipDoubleComplex * v_d, 
    double * w_u,
    double * w_d,
    hipDoubleComplex * H_d,
    hipDoubleComplex * H_SI,
    hipDoubleComplex * H1,
    double p,
    int M,
    int use){


    hipDoubleComplex * X, * Y, * H, *w;
    hipMalloc((void**)&X, M*M*sizeof(hipDoubleComplex));
    hipMalloc((void**)&Y, M*use*sizeof(hipDoubleComplex));
    hipMalloc((void**)&H, M*M*sizeof(hipDoubleComplex));
    hipMalloc((void**)&w, M*M*sizeof(hipDoubleComplex));


    dim3 block(32,32);
    XY_update<<<3, block>>>(w,H,Y,v_u,v_d,w_u,w_d,H_d,use);

    hipDoubleComplex w_h[1024],test[1024],test2[1024];

    // for(int i=0;i<32;i++){
    //     printf("%d: %f %f\n",i,test[i].x,test[i].y);
    // }
    // for(int i=0;i<64;i++){
    //     printf("%d: %f %f\n",i,test[i].x,test[i].y);
    // }
    
    


    //X Y TODO:可以使用stream
    hipblasHandle_t cublasH;
    hipblasCreate(&cublasH);
    hipDoubleComplex alpha = {1.0,0.0};
    hipDoubleComplex beta1 ={0.0,0.0};
    hipDoubleComplex beta2 ={1.0,0.0};

    hipblasZgemm(
        cublasH,HIPBLAS_OP_C,HIPBLAS_OP_N,
        M,M,M,&alpha,
        H_SI, M,
        w, M,
        &beta1,
        X, M
    );

    
    hipblasZgemm(
        cublasH, HIPBLAS_OP_N,HIPBLAS_OP_N,
        M,M,M, &alpha,
        X,M,
        H_SI,M,
        &beta2,
        H,M
    );


    double temp = 0.5/p+0.08;
    mat_add_I<<<1,block>>>(H,temp);

    hipMemcpy(test,H,M*M*sizeof(hipDoubleComplex),hipMemcpyDeviceToHost);
    for(int i=0;i<65;i++){
        printf("F %d: %f %f\n",i,test[i].x,test[i].y);
    }

    std::ofstream file("x.txt");


    for(int i=0;i<M;i++){
        for(int j=0;j<M;j++){
            double x,y;
            char c1,c2;
            file << test[j*M+i].x << " + " << test[j*M+i].y << "i ";
        }
        file << std::endl;
    }
    file.close();

    hipDoubleComplex * F_inv,*FF;
    hipMalloc((void**)&F_inv, M*M*sizeof(hipDoubleComplex));
    hipMalloc((void**)&FF, M*M*sizeof(hipDoubleComplex));
    hipMemcpy(FF,H,M*M*sizeof(hipDoubleComplex),hipMemcpyDeviceToDevice);
    
    inv(H, F_inv, M);



    hipDoubleComplex test5[1024];
    hipMemcpy(test5,F_inv,M*M*sizeof(hipDoubleComplex),hipMemcpyDeviceToHost);
    for(int i=0;i<65;i++){
        printf("F_inv %d: %f %f\n",i,test5[i].x,test5[i].y);
    }



    hipDoubleComplex * E;
    hipMalloc((void**)&E,M*M*sizeof(hipDoubleComplex));
    hipblasZgemm(
        cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,
        M,M,M,&alpha,
        FF,M,
        F_inv,M,
        &beta1,
        E,M
    );
    hipMemcpy(test2,E,M*M*sizeof(hipDoubleComplex),hipMemcpyDeviceToHost);
    for(int i=0;i<64;i++){
        printf("E %d: %f %f\n",i,test2[i].x,test2[i].y);
    }

    return FF;




    // float l = 0.5 / p;
    // hipComplex alpha2 = {l,0.0};
    // float N = 8;

    // hipblasCgemm(
    //     cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,
    //     M, use, N, &alpha2,
    //     F_RF, M,
    //     F_BB, N,
    //     &beta2,
    //     Y,M
    // );
    
    // hipMemcpy(test2,Y,M*use*sizeof(hipComplex),hipMemcpyDeviceToHost);
    // for(int i=0;i<32;i++){
    //     printf("%d: %f %f\n",i,test2[i].x,test2[i].y);
    // }

    // hipblasCgemm(
    //     cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,
    //     M,use,M,&alpha,
    //     F_inv,M,
    //     Y,M,
    //     &beta1,
    //     F,M
    // );

}


//F_BB_update

//V' * (S^-1)'
__global__ void v_mul_s(hipDoubleComplex * V ,double * S,hipDoubleComplex * VS,int M,int N){
    int m = threadIdx.x;
    int n = threadIdx.y;
    if(m < N&& n < N){
        VS[n * N + m] = hipCmul(hipConj(V[m*N+n]),make_hipDoubleComplex(1.0/S[n],0.0));
    }else{
        VS[n * N + m] = make_hipDoubleComplex(0.0,0.0);
    }
}

void F_BB_update(
    hipDoubleComplex * F_RF,
    hipDoubleComplex * F,
    hipDoubleComplex * F_BB,
    int use){

    int F_RF_m = 32;
    int F_RF_n = 8;

    double *S;
    hipDoubleComplex * U, * V;
    hipMalloc((void**)&S,F_RF_m*sizeof(double));
    hipMalloc((void**)&U,F_RF_m*F_RF_m*sizeof(hipDoubleComplex));
    hipMalloc((void**)&V,F_RF_n*F_RF_n*sizeof(hipDoubleComplex));
    
    
    int lwork;
    hipDoubleComplex * work;
    double * rwork = nullptr;
    int *devInfo = nullptr;
    hipsolverHandle_t cusolverH;
    hipsolverDnCreate(&cusolverH);

    hipsolverDnZgesvd_bufferSize(cusolverH,F_RF_m,F_RF_n,&lwork);
    hipMalloc((void**)&work,lwork*sizeof(hipDoubleComplex));
    
    //SVD
    signed char jobu = 'A';
    signed char jobvt = 'A';

    hipsolverDnZgesvd(
        cusolverH,jobu,jobvt,
        F_RF_m,F_RF_n,F_RF,F_RF_m,
        S,
        U,F_RF_m,
        V,F_RF_n,
        work,lwork,rwork,
        devInfo
    );
    
    hipDoubleComplex * VS;
    hipMalloc((void**)&VS, F_RF_n*F_RF_m*sizeof(hipDoubleComplex));
    v_mul_s<<<1,dim3(F_RF_n,F_RF_m)>>>(V,S,VS,F_RF_m,F_RF_n);

    hipDoubleComplex alpha = make_hipDoubleComplex(1.0,0.0);
    hipDoubleComplex beta = make_hipDoubleComplex(0.0,0.0);
    hipDoubleComplex * F_RF_pinv;
    hipMalloc((void**)&F_RF_pinv, F_RF_n*F_RF_m*sizeof(hipDoubleComplex));

    
    hipblasHandle_t cublasH;
    hipblasCreate(&cublasH);
    hipblasZgemm(
        cublasH,HIPBLAS_OP_N,HIPBLAS_OP_C,
        F_RF_n,F_RF_m,F_RF_m,
        &alpha,
        VS,F_RF_n,
        U,F_RF_m,
        &beta, 
        F_RF_pinv,F_RF_n
    );
    
    hipblasZgemm(
        cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,
        F_RF_n,use,F_RF_m,
        &alpha,
        F_RF_pinv,F_RF_n,
        F,F_RF_m,
        &beta,
        F_BB,F_RF_n
    );

}

//F_RF_update

__global__ void RF_update(hipComplex * F_RF, hipComplex * A ,hipComplex *B){
    int k = blockIdx.x;
    int n = threadIdx.x;
    int M = 32;
    int N = 8;
    __shared__ hipComplex V[32];
    __shared__ hipComplex x[16];
    if(n < N){
        V[n] = F_RF[n*M+k]; 
    }

    for(int s=0;s<N;s++){
        if(n < N){
            x[n] = hipCmulf(make_hipComplex(-1.0*V[n].x,-1.0*V[n].y),A[s*N+n] );
        }
        else if(n == N){
            x[n] = B[s*N+k];
        }
        else if(n == N+1){
            x[n] = hipCmulf(V[s],A[s*N+s]);
        }
        else{
            x[n] = make_hipComplex(0.0,0.0);
        }
        __syncthreads();

        for(int i = 8; i > 0 ; i >>= 1){
            if(n < i)
                x[n] = hipCaddf(x[n], x[n+i]);
            __syncthreads();
        }
        __syncthreads();
        if(n==0)
            V[s] = hipCdivf(x[0],make_hipComplex(hipCabsf(x[0]),0.0));
        __syncthreads();
    }
    if(n < N){
        F_RF[n*M+k] = V[n] ; 
    }

}


void F_RF_update(
    hipblasHandle_t cublasH,
    hipComplex* F_BB,
    hipComplex* F,
    hipComplex* F_RF,
    int use){

    int m = 32;
    int n = 8;

    hipComplex * A, *B;
    hipMalloc((void**)&A, n*n*sizeof(hipComplex));
    hipMalloc((void**)&B, m*n*sizeof(hipComplex));

    hipComplex alpha = {1.0,0.0};
    hipComplex beta = {0.0,0.0};
    hipblasCgemm(
        cublasH,
        HIPBLAS_OP_N,HIPBLAS_OP_C,
        n,n,use,
        &alpha,
        F_BB,n,
        F_BB,n,
        &beta,
        A,n
    );
    hipblasCgemm(
        cublasH,
        HIPBLAS_OP_N,HIPBLAS_OP_C,
        m,n,use,
        &alpha,
        F,m,
        F_BB,n,
        &beta,
        B,m
    );

    RF_update<<<m,m>>>(F_RF, A, B);

}

//V_u V_d update

__global__ void A_update(
    hipComplex * A,
    float N,
    hipComplex * F,
    float * p_u,
    hipComplex * H_u,
    hipComplex * H_SI,
    hipComplex * I_W2B,
    int use){

    int k = blockIdx.x;
    int m = threadIdx.x;
    int n = threadIdx.y;

    int M = 32;

    __shared__ hipComplex A_s[1024];
    if (k < use) {
        hipComplex a;
        a = hipCmulf(H_u[k*M+m], hipConjf(H_u[k*M+n]));
        A_s[n*M+m] = hipCmulf(make_hipComplex(p_u[k], 0.0), a);
    }
    else if (k <= use*2) {
        __shared__ hipComplex H_v[32];
        int k_u = k - use;
        if(m == 0 && n < M){
            hipComplex a = {0.0,0.0};
            for(int i=0; i<M; i++){
                a = hipCaddf(a, hipCmulf(H_SI[i*M+n], F[k*M+i]));
            }
            H_v[n] = a;
        }
        __syncthreads();

        A_s[n*M+m] = hipCmulf(H_v[m],H_v[n]);
    }

    atomicAdd(&A[n*M+m].x, A_s[n*M+m].x);
    atomicAdd(&A[n*M+m].y, A_s[n*M+m].y);

    if(k == 0){
        A[n*M+m] = hipCaddf(A[n*M+m], I_W2B[n*M+m]);
        if(m == n){
            A[n*M+m] = hipCaddf(A[n*M+m], make_hipComplex(N, 0.0));
        }
    }
}

__global__ void a_update(
    hipComplex * a,
    float N,
    hipComplex * H_V,
    float * p_u,
    hipComplex * H1,
    float * I_W2U,
    int use){
    
    int m = threadIdx.x;

    float a_f = 0;
    for(int i=0;i<use;i++){
        a_f += p_u[i] * pow(hipCabsf(H1[i*use+m]),2);
    }
    
    hipComplex aa;
    aa = hipCaddf(make_hipComplex(N, 0.0), H_V[m*use+m]);
    a_f = a_f + I_W2U[m];
    a[m] = hipCaddf(aa, make_hipComplex(a_f, 0.0));

}

__global__ void B_update(
    hipComplex * B,
    hipComplex * H_u,
    float * p_u){

    int m = threadIdx.x;
    int n = threadIdx.y;
    int M = 32;

    B[n*M+m] = hipCmulf(H_u[n*M+m],make_hipComplex(sqrt(p_u[n]),0.0));


}

__global__ void VE_update(
    hipComplex * v_u,
    hipComplex * v_d,
    hipComplex * A,
    hipComplex * A_inv,
    hipComplex * B,
    hipComplex * a,
    hipComplex * b,
    float * e_u,
    float * e_d,
    int use){
    int k = blockIdx.x;
    int m = threadIdx.x;
    int n = threadIdx.y;
    int M = 32;

    if(k < use){
        __shared__ hipComplex v_u_l[32];
        if(m == 0){
            hipComplex vv = {0.0,0.0};
            for(int i=0;i<M;i++){
                vv = hipCaddf(vv, hipCmulf(A_inv[i*M+n], B[k*M+i]));
            }
            v_u_l[n] = vv;
            v_u[k*M+n] = vv;
        }
        __syncthreads();

        __shared__ float E_u[1024];
        E_u[n*M+m] = hipCmulf(hipConjf(v_u_l[m]), hipCmulf(A[n*M+m], v_u_l[n])).x;

        __syncthreads();

        if(m == 0){
            E_u[n*M+m] = E_u[n*M+m] - 2 * hipCmulf(hipConjf(v_u_l[n]), B[k*M+n]).x;
        }
        __syncthreads();

        int tid = m*M+n;
        for(int i=512;i>0;i>>=1){
            if(tid < i){
                E_u[tid] = E_u[tid] + E_u[tid+i];
            }
            __syncthreads();
        }

        __syncthreads();
        e_u[k] = E_u[0] + 1.0;
    }
    else if(k < use*2){
        if(m == 0 && n == 0){
            hipComplex v_dd;
            v_dd = hipCdivf(b[k*use+k], a[k]);
            v_d[k] = v_dd;
            e_d[k] = a[k].x * pow(hipCabsf(v_dd),2) - 2 * hipCmulf(v_dd,b[k*use+k]).x + 1;
        }
    }

}

void V_update(
    hipblasHandle_t cublasH,
    float N,
    hipComplex * F,
    float * p_u,
    hipComplex * v_u,
    hipComplex * v_d,
    float * e_u,
    float * e_d,
    hipComplex * H_u,
    hipComplex * H_SI,
    hipComplex * H_d,
    hipComplex * H1,
    hipComplex * I_W2B,
    float * I_W2U,
    int use){
    
    int M =32;

    hipComplex *A,*B, *a, *b;
    hipMalloc((void**)&A, M*M*sizeof(hipComplex));

    A_update<<<use*2, dim3(M,M)>>>(A, N, F, p_u, H_u, H_SI, I_W2B, use);

    hipComplex *H_F,*H_F_t;
    hipMalloc((void**)&H_F, use*use*sizeof(hipComplex));
    hipMalloc((void**)&H_F_t, use*use*sizeof(hipComplex));

    hipComplex alpha = {1.0,0.0};
    hipComplex beta = {0.0,0.0};

    hipblasCgemm(
        cublasH,
        HIPBLAS_OP_N,HIPBLAS_OP_N,
        use,use,M,
        &alpha,
        H_d,use,
        F,M,
        &beta,
        H_F,use
    );
    hipblasCgemm(
        cublasH,
        HIPBLAS_OP_N,HIPBLAS_OP_C,
        use,use,use,
        &alpha,
        H_F,use,
        H_F,use,
        &beta,
        H_F_t,use
    );

    a_update<<<1,use>>>(a,N,H_F_t,p_u,H1,I_W2U,use);
    B_update<<<1,dim3(M,use)>>>(B,H_u,p_u);
    hipblasCgemm(
        cublasH,
        HIPBLAS_OP_N,HIPBLAS_OP_N,
        use,use,M,
        &alpha,
        H_d,use,
        F,M,
        &beta,
        b,use
    );

    hipComplex * A_inv;
    hipMalloc((void**)&A_inv, M*M*sizeof(hipComplex));
    //inv(A, A_inv, M);
    
    VE_update<<<use*2, dim3(M,M)>>>(v_u, v_d, A, A_inv, B, a, b, e_u, e_d, use);
}


//计算f

__global__ void f_cal( 
    float * f,   
    float * w_u,
    float * w_d,
    float * e_u,
    float * e_d,
    float F_norm,
    int p,
    int use){
    
    int m = threadIdx.x;
    int n = threadIdx.y;

    __shared__ float sh[1024];
    if(m == 0){
        sh[m*use+n] = w_u[n] * e_u[n];
    }
    else if (m == 1) {
        sh[m*use+n] = -1.0 * __log2f(w_u[n]);
    }
    else if (m == 2) {
        sh[m*use+n] = w_d[n] * e_d[n];
    }
    else if (m == 3) {
        sh[m*use+n] = -1.0 * __log2f(w_d[n]);
    }
    __syncthreads();

    int tid = m*use+n;
    for (int i = 2*use; i > 0; i >>= 1)
    {
        if(tid < i){
            sh[tid] = sh[tid] + sh[tid+i];
        }
        __syncthreads();
        /* code */
    }
    __syncthreads();
    if(tid == 0)
        f[0] = sh[0] + (0.5 / p) * F_norm * F_norm; 
    
}

float F_cal(
    hipblasHandle_t cublasH,
    float * w_u,
    float * w_d,
    float * e_u,
    float * e_d,
    hipComplex * F,
    hipComplex * F_RF,
    hipComplex * F_BB,
    int use,
    int p,
    int RF){
    
    int M = 32;

    hipComplex * F_gap;
    float F_norm;
    hipMalloc((void**)&F_gap, M*use*sizeof(hipComplex));
    hipMemcpy(F_gap, F, M*use*sizeof(hipComplex), hipMemcpyDeviceToDevice);
    hipComplex alpha = {-1.0,0.0};
    hipComplex beta = {1.0,0.0};
    
    hipblasCgemm(
        cublasH,
        HIPBLAS_OP_N,HIPBLAS_OP_N,
        M,use,RF,
        &alpha,
        F_RF, M,
        F_BB, RF,
        &beta,
        F_gap,M
    );

    hipblasScnrm2(
        cublasH, M*use,
        F_gap, 1, &F_norm
    );
    float * f;
    hipMalloc((void **)&f,sizeof(float));
    f_cal<<<1,dim3(4,use)>>>(f, w_u, w_d, e_u, e_d, F_norm, p, use);

    float f_h;
    hipMemcpy(&f_h, f, sizeof(float), hipMemcpyDeviceToHost);
    return f_h;

}


// int main() {

//     double e_u[ 4 ] = { 1,2,3,4 };
//     double e_d[ 4 ] = { 1,2,3,4 };

//     int ul_use = 4;

//     double* e_u_d, * e_d_d;
//     hipMalloc((void**)&e_d_d, 4 * sizeof(double));
//     hipMalloc((void**)&e_u_d, 4 * sizeof(double));

//     hipMemcpy(e_u_d, e_u, ul_use * sizeof(double), hipMemcpyHostToDevice);
//     hipMemcpy(e_d_d, e_d, ul_use * sizeof(double), hipMemcpyHostToDevice);


//     double* sigma_u, * sigma_d;
//     hipMalloc((void**)&sigma_u, 4 * sizeof(double));
//     hipMalloc((void**)&sigma_d, ul_use * sizeof(double));

//     printf("%f", e_u[ 0 ]);

//     dim3 blocksize(1, 1, 8);
//     w_updata << < 1, blocksize >> > (sigma_u, sigma_d, e_u_d, e_d_d, ul_use);
//     hipDeviceSynchronize();



//     return 0;
// }