#include "hip/hip_runtime.h"
#include <complex>
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hipsolver.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <fstream>

#include "cu_pdd.h"

  #if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600
  static __inline__ __device__ double atomicAdd(double *address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    if (val==0.0)
      return __longlong_as_double(old);
    do {
      assumed = old;
      old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val +__longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
  }
  #endif



__device__ hipDoubleComplex global_v[256] = {0,0.0};



__global__ void w_update(
    double * w_u, 
    double * w_d, 
    double * e_u, 
    double * e_d, 
    double * A,
    int use) {
    int idx = threadIdx.x;
    if (idx < use) {
        w_u[ idx ] = 1.0 / (log(2.0) * e_u[ idx ]);
        A[idx] = 0.0;
    }
    else if (idx < 2 * use) {
        w_d[ idx - use ] = 1.0 / (log(2.0)  * e_d[ idx - use ]);
    }
}






__global__ void p_update(
    double* p_u, 
    double* w_u,
    double* w_d,
    hipDoubleComplex* v_u,
    hipDoubleComplex* v_d,
    hipDoubleComplex* H_u,
    hipDoubleComplex* H1, 
    hipDoubleComplex* H3,
    double* A,  //[a1,a1 .. ]
    double I_th,
    int M,
    int use)
    {
    //<<<(4,4);32>>>
    int k = blockIdx.x;
    int s = blockIdx.y;
    int n = threadIdx.x;



    __shared__ hipDoubleComplex H_H[32];
    H_H[n] = hipCmul(hipConj(v_u[s * M + n]) , H_u[k * M + n]);
    
    __syncthreads();

    
    for (size_t i = M/2; i > 0; i >>= 1)
    {
        if(n < i){
            H_H[n] = hipCadd(H_H[n] , H_H[n+i]);
        }
        /* code */
        __syncthreads();
    }
    __syncthreads();

    if(n == 0){
        H_H[0] = hipCmul(make_hipDoubleComplex(w_u[s], 0.0), hipCmul(H_H[0], hipConj(H_H[0])));
        
        hipDoubleComplex hh = hipCmul(v_d[s], H1[k*use+s]);
        hh = hipCmul(make_hipDoubleComplex(w_d[s], 0.0), hipCmul(hh, hipConj(hh)));
        hh = hipCadd(hh, H_H[0]);

        atomicAdd(&A[k], hh.x);
    }
    __syncthreads();

    if(s == 0){
        __shared__ double B[32];

        hipDoubleComplex b=hipCmul(hipConj(v_u[k*M+n]), H_u[k*M+n]);
        B[n] = b.x;
        __syncthreads();

        for (size_t i = M/2; i > 0; i >>= 1)
        {
            if(n < i){
                B[n] = B[n] + B[n + i];
            }
            __syncthreads();
            /* code */
        }
        __syncthreads();
        if(n == 0){
            double bb = B[0]*w_u[k];
            double a = A[k];

            if(bb >= 0){
                double p_use = 1.0;
                double b_a = pow(bb / a,2);
                double I_h3 = I_th / hipCreal(hipCmul(H3[k], hipConj(H3[k])));

                p_u[k] = min(min(p_use, b_a) , I_h3);
            }
            else
                p_u[k] = 0;
        }


    }



}

// __global__ void p_update(
//     double* p_u, 
//     double* w_u,
//     double* w_d,
//     hipDoubleComplex* v_u,
//     hipDoubleComplex* v_d,
//     hipDoubleComplex* H_u,
//     hipDoubleComplex* H1, 
//     hipDoubleComplex* H3,
//     double I_th,
//     int M,
//     int use)
//     {
//     //<<<(4,4);32>>>
//     int k = blockIdx.x;
//     int s = blockIdx.y;
//     int n = threadIdx.x;

//     global_v[n] = {0.0,0.0};


//     __shared__ hipDoubleComplex H_H[32];
//     H_H[n] = hipCmul(hipConj(v_u[s * M + n]) , H_u[k * M + n]);
    

//     __syncthreads();

    
//     for (size_t i = M/2; i > 0; i >>= 1)
//     {
//         if(n < i){
//             H_H[n] = hipCadd(H_H[n] , H_H[n+i]);
//         }
//         /* code */
//         __syncthreads();
//     }
//     __syncthreads();

//     if(n == 0){
//         H_H[0] = hipCmul(make_hipDoubleComplex(w_u[s], 0.0), hipCmul(H_H[0], hipConj(H_H[0])));
        
//         hipDoubleComplex hh = hipCmul(v_d[s], H1[k*use+s]);
//         hh = hipCmul(make_hipDoubleComplex(w_d[s], 0.0), hipCmul(hh, hipConj(hh)));
//         hh = hipCadd(hh, H_H[0]);
//         global_v[k*use+s] = hh;
//     }
//     __syncthreads();
//     __shared__ double B[32];

//     hipDoubleComplex b;
//     b = hipCmul(hipConj(v_u[k*M+n]), H_u[k*M+n]);
//     B[n] = b.x;
//     __syncthreads();

//     for (size_t i = M/2; i > 0; i >>= 1)
//     {
//         if(n < i){
//             B[n] = B[n] + B[n + i];
//         }
//         __syncthreads();
//         /* code */
//     }
//     __syncthreads();


//     double bb = B[0]*w_u[k];


//     if(s == 0 && n == 0){

//         hipDoubleComplex a = {0.0,0.0};
//         for(int i=0;i<use;i++){
//             a = hipCadd(a, global_v[k*use+i]);
//         }

//         hipDoubleComplex b = make_hipDoubleComplex(bb, 0.0);

//         if(b.x >= 0){
//             double p_use = 1.0;
//             double b_a = pow(hipCabs(hipCdiv(b , a)), 2);
//             double I_h3 = I_th / hipCreal(hipCmul(H3[k], hipConj(H3[k])));

//             p_u[k] = min(min(p_use, b_a) , I_h3);
//         }
//         else
//             p_u[k] = 0;

//     }
//     __syncthreads();

// }


//F_update
//按照列存储运算，输出参数为列存储
__global__ void XY_update(
    hipDoubleComplex * X,
    hipDoubleComplex * H,
    hipDoubleComplex * Y, 
    hipDoubleComplex * v_u, 
    hipDoubleComplex * v_d, 
    double * w_u,
    double * w_d,
    hipDoubleComplex * H_d,
    int use){
    
    int k = blockIdx.x;
    int m = threadIdx.x;
    int n = threadIdx.y;
    int M = 32;

    if(k == 0){
        __shared__ hipDoubleComplex w[1024];
        w[n*M+m] = {0.0,0.0};
        hipDoubleComplex ww;
        for(int i=0;i<use;i++){
            ww = hipCmul(v_u[i*M+m],hipConj(v_u[i*M+n]));
            ww = hipCmul(ww, make_hipDoubleComplex(w_u[i],0.0));
            w[n*M+m] = hipCadd(w[n*M+m], ww);
            __syncthreads();
        }
        X[n*M+m] = w[n*M+m];
        if (m == n) {
            X[n*M+m].y = 0.0;
        }
    }
    else if (k == 1) {
        __shared__ hipDoubleComplex h[1024];
        h[n*M+m] = {0.0,0.0};
        hipDoubleComplex hh;
        for(int i=0;i<use;i++){
            hh = hipCmul(hipConj(v_d[i]), v_d[i]);
            hh = hipCmul(hipConj(H_d[m*use+i]), hh);
            hh = hipCmul(hh, H_d[n*use+i]);
            h[n*M+m] = hipCadd(h[n*M+m], hipCmul(make_hipDoubleComplex(w_d[i], 0.0), hh));
            __syncthreads();
        }
        H[n*M+m] = h[n*M+m];
    }
    else {
        if (m < 32 && n < use) {
            hipDoubleComplex yy;
            yy = hipCmul(hipConj(H_d[m*use+n]), hipConj(v_d[n]));
            Y[n*M+m] = hipCmul(make_hipDoubleComplex(w_d[n],0.0), yy);
        }
    }
      
}

__global__ void mat_add_I(hipDoubleComplex * x, double p){
    int m = threadIdx.x;
    int n = threadIdx.y;


    int M = 32;
    if(m == n){
        x[m*M+n] = hipCadd(x[m*M+n], make_hipDoubleComplex(p, 0.0));
    }
}



__global__ void fun6(hipDoubleComplex*A,int M){
    int n = threadIdx.x;
    A[n*M+n] = make_hipDoubleComplex(1.0, 0.0);
}



void inv(hipDoubleComplex * F, hipDoubleComplex * F_inv, int M,hipsolverHandle_t cusolverH){

    int lwork;
    hipDoubleComplex * work;
    int *devIpiv = nullptr;
    int *devInfo = nullptr;
    hipsolverDnZgetrf_bufferSize(cusolverH,M,M,F,M,&lwork);
    
    hipMalloc((void**)&work,lwork*sizeof(hipDoubleComplex));
    hipMalloc((void**)&devIpiv, M * sizeof(int));
    hipsolverDnZgetrf(cusolverH, M, M, F, M, work, NULL, devInfo);

    fun6<<<1,M>>>(F_inv,M);

    hipsolverDnZgetrs(cusolverH, HIPBLAS_OP_N, M, M, F, M, NULL, F_inv, M, devInfo);
    
    //hipsolverDnDestroy(cusolverH);
    hipFree(work);
    hipFree(devIpiv);
    
}






void F_update(
    hipDoubleComplex * F, 
    hipDoubleComplex * F_RF,
    hipDoubleComplex * F_BB,
    hipDoubleComplex * v_u, 
    hipDoubleComplex * v_d, 
    double * w_u,
    double * w_d,
    hipDoubleComplex * H_d,
    hipDoubleComplex * H_SI,
    hipDoubleComplex * H1,
    hipDoubleComplex * lambda,
    double p,
    int M,
    int use,
    hipblasHandle_t cublasH,
    hipsolverHandle_t cusolverH
    ){


    //hipblasCreate(&cublasH);

    hipDoubleComplex * X, * Y, * H, *w;
    hipMalloc((void**)&X, M*M*sizeof(hipDoubleComplex));
    hipMalloc((void**)&Y, M*use*sizeof(hipDoubleComplex));
    hipMalloc((void**)&H, M*M*sizeof(hipDoubleComplex));
    hipMalloc((void**)&w, M*M*sizeof(hipDoubleComplex));


    dim3 block(32,32);
    XY_update<<<3, block>>>(w,H,Y,v_u,v_d,w_u,w_d,H_d,use);
   
    //hipDoubleComplex test[1024];


    //X Y TODO:可以使用stream

    hipDoubleComplex alpha = {1.0,0.0};
    hipDoubleComplex beta1 ={0.0,0.0};
    hipDoubleComplex beta2 ={1.0,0.0};

    hipblasZgemm(
        cublasH,HIPBLAS_OP_C,HIPBLAS_OP_N,
        M,M,M,&alpha,
        H_SI, M,
        w, M,
        &beta1,
        X, M
    );

    
    hipblasZgemm(
        cublasH, HIPBLAS_OP_N,HIPBLAS_OP_N,
        M,M,M, &alpha,
        X,M,
        H_SI,M,
        &beta2,
        H,M
    );

    //alphe = 0.08
    double temp = 0.5/p+0.08;
    mat_add_I<<<1,block>>>(H,temp);


    hipDoubleComplex * F_inv;
    hipMalloc((void**)&F_inv, M*M*sizeof(hipDoubleComplex));

    // hipMemcpy(test,H,M*M*sizeof(hipDoubleComplex),hipMemcpyDeviceToHost);
    // for(int i=0;i<32*32;i++){
    //     printf("H :%d: %f %f\n",i,test[i].x,test[i].y);
    // }
   
    inv(H, F_inv, M,cusolverH);

    // hipMemcpy(test,F_inv,M*M*sizeof(hipDoubleComplex),hipMemcpyDeviceToHost);
    // for(int i=0;i<32*32;i++){
    //     printf("inv :%d: %f %f\n",i,test[i].x,test[i].y);
    // }

    double l = 0.5 / p;
    hipDoubleComplex alpha2 = {l,0.0};
    hipDoubleComplex beta3 = {-0.5,0.0};

    int N = 8;

    hipblasZgemm(
        cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,
        M, use, N, &alpha2,
        F_RF, M,
        F_BB, N,
        &beta2,
        Y,M
    );

    hipblasZgeam(
        cublasH,
        HIPBLAS_OP_N,HIPBLAS_OP_N,
        M,use,
        &alpha,
        Y,M,
        &beta3,
        lambda,M,
        Y,M
    );
    
    // hipMemcpy(test,Y,M*use*sizeof(hipDoubleComplex),hipMemcpyDeviceToHost);
    // for(int i=0;i<32;i++){
    //     printf("Y :%d: %f %f\n",i,test[i].x,test[i].y);
    // }


    hipblasZgemm(
        cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,
        M,use,M,&alpha,
        F_inv,M,
        Y,M,
        &beta1,
        F,M
    );


    //hipDoubleComplex test[1024];
    // hipMemcpy(test,F,M*use*sizeof(hipDoubleComplex),hipMemcpyDeviceToHost);
    // for(int i=0;i<4;i++){
    //     printf("F :%d: %f %f\n",i,test[i].x,test[i].y);
    // }


}


//F_BB_update

//V' * (S^-1)'
__global__ void v_mul_s(hipDoubleComplex * V ,double * S,hipDoubleComplex * VS,int M,int N){
    int m = threadIdx.x;
    int n = threadIdx.y;
    if(m < N&& n < N){
        VS[n * N + m] = hipCmul(hipConj(V[m*N+n]),make_hipDoubleComplex(1.0/S[n],0.0));
    }else{
        VS[n * N + m] = make_hipDoubleComplex(0.0,0.0);
    }
}

void F_BB_update(
    hipDoubleComplex * F_RF,
    hipDoubleComplex * F,
    hipDoubleComplex * F_BB,
    hipDoubleComplex * lambda,
    double p,
    int M,
    int N,
    int use,
    hipblasHandle_t cublasH,
    hipsolverHandle_t cusolverH){


    double *S;
    hipDoubleComplex * U, * V;
    hipMalloc((void**)&S,M*sizeof(double));
    hipMalloc((void**)&U,M*M*sizeof(hipDoubleComplex));
    hipMalloc((void**)&V,N*N*sizeof(hipDoubleComplex));
    
    
    int lwork;
    hipDoubleComplex * work;
    double * rwork = nullptr;
    int *devInfo = nullptr;

    hipsolverDnZgesvd_bufferSize(cusolverH,M,N,&lwork);
    hipMalloc((void**)&work,lwork*sizeof(hipDoubleComplex));
    
    //SVD
    signed char jobu = 'A';
    signed char jobvt = 'A';

    hipDoubleComplex * F_RF_;
    hipMalloc((void**)&F_RF_,M*N*sizeof(hipDoubleComplex));
    hipMemcpy(F_RF_,F_RF,M*N*sizeof(hipDoubleComplex),hipMemcpyDeviceToDevice);

    hipsolverDnZgesvd(
        cusolverH,jobu,jobvt,
        M,N,F_RF_,M,
        S,
        U,M,
        V,N,
        work,lwork,rwork,
        devInfo
    );
    
    hipDoubleComplex * VS;
    hipMalloc((void**)&VS, N*M*sizeof(hipDoubleComplex));
    v_mul_s<<<1,dim3(N,M)>>>(V,S,VS,M,N);

    hipDoubleComplex alpha = make_hipDoubleComplex(1.0,0.0);
    hipDoubleComplex beta = make_hipDoubleComplex(0.0,0.0);
    hipDoubleComplex beta_p = {p,0.0};

    hipDoubleComplex * F_RF_pinv;
    hipMalloc((void**)&F_RF_pinv, N*M*sizeof(hipDoubleComplex));

    
    hipblasZgemm(
        cublasH,HIPBLAS_OP_N,HIPBLAS_OP_C,
        N,M,M,
        &alpha,
        VS,N,
        U,M,
        &beta, 
        F_RF_pinv,N
    );
    
    hipDoubleComplex * C;
    hipMalloc((void**)&C,M*use*sizeof(hipDoubleComplex));
    hipblasZgeam(
        cublasH,
        HIPBLAS_OP_N,HIPBLAS_OP_N,
        M,use,
        &alpha,
        F,M,
        &beta_p,
        lambda,M,
        C,M
    );

    hipblasZgemm(
        cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,
        N,use,M,
        &alpha,
        F_RF_pinv,N,
        C,M,
        &beta,
        F_BB,N
    );

    // hipDoubleComplex test[1024];
    // hipMemcpy(test,F_BB,N*use*sizeof(hipDoubleComplex),hipMemcpyDeviceToHost);
    // for(int i=0;i<4;i++){
    //     printf("F_BB :%d: %f %f\n",i,test[i].x,test[i].y);
    // }
}

//F_RF_update

__global__ void RF_update(
    hipDoubleComplex * F_RF, 
    hipDoubleComplex * A ,
    hipDoubleComplex * B,
    int M,
    int N){
    //<<<M,M>>>
    int k = blockIdx.x;
    int n = threadIdx.x;

    __shared__ hipDoubleComplex V[32];
    __shared__ hipDoubleComplex x[32];
    if(n < N){
        V[n] = F_RF[n*M+k]; 
    }

    for(int s=0;s<N;s++){
        
        x[n] = make_hipDoubleComplex(0.0, 0.0);
        __syncthreads();
        if(n < N){
            x[n] = hipCmul(make_hipDoubleComplex(-1.0*V[n].x,-1.0*V[n].y),A[s*N+n] );
        }
        else if(n == N){
            x[n] = B[s*M+k];
        }
        else if(n == N+1){
            x[n] = hipCmul(V[s],A[s*N+s]);
        }
        else{
            x[n] = make_hipDoubleComplex(0.0,0.0);
        }
        __syncthreads();

        for(int i = M/2; i > 0 ; i >>= 1){
            if(n < i)
                x[n] = hipCadd(x[n], x[n+i]);
            __syncthreads();
        }
        __syncthreads();
        if(n==0){
            V[s] = hipCdiv(x[0],make_hipDoubleComplex(hipCabs(x[0]),0.0));
        
        }
        __syncthreads();


    }
    if(n < N){
        F_RF[n*M+k] = V[n] ; 
    }

}


void F_RF_update(
    hipDoubleComplex* F_BB,
    hipDoubleComplex* F,
    hipDoubleComplex* F_RF,
    hipDoubleComplex* lambda,
    double p,
    int M,
    int N,
    int use,
    hipblasHandle_t cublasH){

    hipDoubleComplex * A, *B, *C;
    hipMalloc((void**)&A, N*N*sizeof(hipDoubleComplex));
    hipMalloc((void**)&B, M*N*sizeof(hipDoubleComplex));
    hipMalloc((void**)&C, M*use*sizeof(hipDoubleComplex));

    hipDoubleComplex alpha = {1.0,0.0};
    hipDoubleComplex beta = {0.0,0.0};
    hipDoubleComplex beta_p = {p,0.0};
    hipblasZgemm(
        cublasH,
        HIPBLAS_OP_N,HIPBLAS_OP_C,
        N,N,use,
        &alpha,
        F_BB,N,
        F_BB,N,
        &beta,
        A,N
    );
    
    hipblasZgeam(
        cublasH,
        HIPBLAS_OP_N,HIPBLAS_OP_N,
        M,use,
        &alpha,
        F,M,
        &beta_p,
        lambda,M,
        C,M
    );

    hipblasZgemm(
        cublasH,
        HIPBLAS_OP_N,HIPBLAS_OP_C,
        M,N,use,
        &alpha,
        C,M,
        F_BB,N,
        &beta,
        B,M
    );
    //hipDoubleComplex test[1024];


    RF_update<<<M,M>>>(F_RF, A, B, M, N);

    // hipMemcpy(test,F_RF,M*N*sizeof(hipDoubleComplex),hipMemcpyDeviceToHost);
    // for(int i=0;i<4;i++){
    //     printf("F_RF :%d: %f %f\n",i,test[i].x,test[i].y);
    // }

}

//V_u V_d update




__global__ void A_update(
    hipDoubleComplex * A,
    double N,
    hipDoubleComplex * F,
    double * p_u,
    hipDoubleComplex * H_u,
    hipDoubleComplex * H_SI,
    hipDoubleComplex * I_W2B,
    int use){

    int m = threadIdx.x;
    int n = threadIdx.y;
    //<<<1,(M,M)>>>
    int M = 32;

    __shared__ hipDoubleComplex A_s[1024];
    __shared__ hipDoubleComplex H_v[32];

    A_s[n*M+m] = {0.0,0.0};

    for(int k=0;k<use;k++){
        hipDoubleComplex a;
        a = hipCmul(H_u[k*M+m], hipConj(H_u[k*M+n]));
        a = hipCmul(make_hipDoubleComplex(p_u[k], 0.0), a);

        // if( n==0 ){
        //     printf("a :%d %d %e %e\n",k ,m,a.x,a.y);
        // }
        __syncthreads();
        if(m == 0){
            hipDoubleComplex aa = {0.0,0.0};
            for(int i=0; i<M; i++){
                aa = hipCadd(aa, hipCmul(H_SI[i*M+n], F[k*M+i]));
            }
            H_v[n] = aa;

            // printf("a :%d %d %e %e\n",k ,n,H_v[n].x,H_v[n].y);
        }
        __syncthreads();

        // if(n==0){
        //     printf("A_s_pre %d :%e %e\n",k, A_s[n*M+m].x,A_s[n*M+m].y);
        // }
        a = hipCadd(a, A_s[n*M+m]);
        A_s[n*M+m] = hipCadd(a, hipCmul(H_v[m],hipConj(H_v[n])));
        // if(n==0){
        //     printf("A_s %d :%e %e\n",k, A_s[n*M+m].x,A_s[n*M+m].y);
        // }
        __syncthreads();

    }
    __syncthreads();


    A_s[n*M+m] = hipCadd(A_s[n*M+m], I_W2B[n*M+m]);
    __syncthreads();
    if(m == n){
        A_s[n*M+m] = hipCadd(A_s[n*M+m], make_hipDoubleComplex(N, 0.0));
    }
    __syncthreads();
    A[n*M+m] = A_s[n*M+m];

}

// __global__ void A_update(
//     hipDoubleComplex * A,
//     double N,
//     hipDoubleComplex * F,
//     double * p_u,
//     hipDoubleComplex * H_u,
//     hipDoubleComplex * H_SI,
//     hipDoubleComplex * I_W2B,
//     int use){

//     int k = blockIdx.x;
//     int m = threadIdx.x;
//     int n = threadIdx.y;

//     int M = 32;

//     __shared__ hipDoubleComplex A_s[1024];
//     if (k < use) {
//         hipDoubleComplex a;
//         a = hipCmul(H_u[k*M+m], hipConj(H_u[k*M+n]));
//         A_s[n*M+m] = hipCmul(make_hipDoubleComplex(p_u[k], 0.0), a);
//     }
//     else if (k <= use*2) {
//         __shared__ hipDoubleComplex H_v[32];
//         int k_u = k - use;
//         if(m == 0 && n < M){
//             hipDoubleComplex a = {0.0,0.0};
//             for(int i=0; i<M; i++){
//                 a = hipCadd(a, hipCmul(H_SI[i*M+n], F[k_u*M+i]));
//             }
//             H_v[n] = a;
//         }
//         __syncthreads();

//         A_s[n*M+m] = hipCmul(H_v[m],hipConj(H_v[n]));
//     }

//     atomicAdd(&A[n*M+m].x, A_s[n*M+m].x);
//     atomicAdd(&A[n*M+m].y, A_s[n*M+m].y);

//     if(k == 0){
//         A[n*M+m] = hipCadd(A[n*M+m], I_W2B[n*M+m]);
//         if(m == n){
//             A[n*M+m] = hipCadd(A[n*M+m], make_hipDoubleComplex(N, 0.0));
//         }
//     }
// }



__global__ void a_update(
    hipDoubleComplex * a,
    double N,
    hipDoubleComplex * H_V,
    double * p_u,
    hipDoubleComplex * H1,
    double * I_W2U,
    int use){
    
    int m = threadIdx.x;

    double a_f = 0;
    for(int i=0;i<use;i++){
        a_f += p_u[i] * pow(hipCabs(H1[i*use+m]),2);
    }
    
    hipDoubleComplex aa;
    aa = hipCadd(make_hipDoubleComplex(N, 0.0), H_V[m*use+m]);
    a_f = a_f + I_W2U[m];
    a[m] = hipCadd(aa, make_hipDoubleComplex(a_f, 0.0));

}

__global__ void B_update(
    hipDoubleComplex * B,
    hipDoubleComplex * H_u,
    double * p_u){

    int m = threadIdx.x;
    int n = threadIdx.y;
    int M = 32;

    B[n*M+m] = hipCmul(H_u[n*M+m],make_hipDoubleComplex(sqrt(p_u[n]),0.0));


}

__global__ void VE_update(
    hipDoubleComplex * v_u,
    hipDoubleComplex * v_d,
    hipDoubleComplex * A,
    hipDoubleComplex * A_inv,
    hipDoubleComplex * B,
    hipDoubleComplex * a,
    hipDoubleComplex * b,
    double * e_u,
    double * e_d,
    int use){
    int k = blockIdx.x;
    int m = threadIdx.x;
    int n = threadIdx.y;
    int M = 32;
    __shared__ hipDoubleComplex v_u_l[32];
    if(n == 0)
        v_u_l[m] = {0.0,0.0};
    __syncthreads();
    if(k < use){
        if(m == 0){
            hipDoubleComplex vv = {0.0,0.0};
            for(int i=0;i<M;i++){
                vv = hipCadd(vv, hipCmul(A_inv[i*M+n], B[k*M+i]));
            }
            v_u_l[n] = vv;
            v_u[k*M+n] = vv;
        }
        __syncthreads();

        __shared__ double E_u[1024];
        E_u[n*M+m] = 0.0;
        E_u[n*M+m] = hipCmul(hipConj(v_u_l[m]), hipCmul(A[n*M+m], v_u_l[n])).x;

        __syncthreads();

        if(m == 0){
            E_u[n*M+m] = E_u[n*M+m] - 2 * hipCmul(hipConj(v_u_l[n]), B[k*M+n]).x;
        }
        __syncthreads();

        int tid = m*M+n;
        for(int i=512;i>0;i>>=1){
            if(tid < i){
                E_u[tid] = E_u[tid] + E_u[tid+i];
            }
            __syncthreads();
        }

        __syncthreads();
        if(n==0&&m==0)
            e_u[k] = E_u[0] + 1.0;
    }
    else if(k < use*2){
        if(m == 0 && n == 0){
            int kk = k - use;
            hipDoubleComplex v_dd;
            v_dd = hipCdiv(b[kk*use+kk], a[kk]);
            v_d[kk] = v_dd;
            e_d[kk] = a[kk].x * pow(hipCabs(v_dd),2) - 2 * hipCmul(v_dd,b[kk*use+kk]).x + 1;
        }
    }

}

void V_update(
    double N,
    hipDoubleComplex * F,
    double * p_u,
    hipDoubleComplex * v_u,
    hipDoubleComplex * v_d,
    double * e_u,
    double * e_d,
    hipDoubleComplex * H_u,
    hipDoubleComplex * H_SI,
    hipDoubleComplex * H_d,
    hipDoubleComplex * H1,
    hipDoubleComplex * I_W2B,
    double * I_W2U,
    int use,
    hipblasHandle_t cublasH,
    hipsolverHandle_t cusolverH){
    
    int M =32;

    //test
    //hipDoubleComplex test[1024];
    //double test2[1024];

    hipDoubleComplex *A,*B, *a, *b;
    hipMalloc((void**)&A, M*M*sizeof(hipDoubleComplex));
    hipMalloc((void**)&B, M*use*sizeof(hipDoubleComplex));
    hipMalloc((void**)&a, use*sizeof(hipDoubleComplex));
    hipMalloc((void**)&b, use*sizeof(hipDoubleComplex));

    A_update<<<1, dim3(M,M)>>>(A, N, F, p_u, H_u, H_SI, I_W2B, use);


    // hipMemcpy(test,A,M*M*sizeof(hipDoubleComplex),hipMemcpyDeviceToHost);
    // for(int i=0;i<4;i++){
    //     printf("A:%d :%e %e\n",i,test[i].x,test[i].y);
    // }


    hipDoubleComplex *H_F,*H_F_t;
    hipMalloc((void**)&H_F, use*use*sizeof(hipDoubleComplex));
    hipMalloc((void**)&H_F_t, use*use*sizeof(hipDoubleComplex));

    hipDoubleComplex alpha = {1.0,0.0};
    hipDoubleComplex beta = {0.0,0.0};

    hipblasZgemm(
        cublasH,
        HIPBLAS_OP_N,HIPBLAS_OP_N,
        use,use,M,
        &alpha,
        H_d,use,
        F,M,
        &beta,
        H_F,use
    );
    hipblasZgemm(
        cublasH,
        HIPBLAS_OP_N,HIPBLAS_OP_C,
        use,use,use,
        &alpha,
        H_F,use,
        H_F,use,
        &beta,
        H_F_t,use
    );

    a_update<<<1,use>>>(a,N,H_F_t,p_u,H1,I_W2U,use);

    B_update<<<1,dim3(M,use)>>>(B,H_u,p_u);

    


    hipblasZgemm(
        cublasH,
        HIPBLAS_OP_N,HIPBLAS_OP_N,
        use,use,M,
        &alpha,
        H_d,use,
        F,M,
        &beta,
        b,use
    );



    hipDoubleComplex * A_inv,*AA;
    hipMalloc((void**)&A_inv, M*M*sizeof(hipDoubleComplex));
    hipMalloc((void**)&AA, M*M*sizeof(hipDoubleComplex));
    hipMemcpy(AA,A,M*M*sizeof(hipDoubleComplex),hipMemcpyDeviceToDevice);

    inv(A, A_inv, M,cusolverH);
    // hipMemcpy(test,AA,M*M*sizeof(hipDoubleComplex),hipMemcpyDeviceToHost);
    // for(int i=0;i<32;i++){
    //     printf("A :%d: %f %f\n",i,test[i].x,test[i].y);
    // }

    // hipMemcpy(test,A_inv,M*M*sizeof(hipDoubleComplex),hipMemcpyDeviceToHost);
    // for(int i=0;i<32;i++){
    //     printf("Ainv :%d: %f %f\n",i,test[i].x,test[i].y);
    // }
    
    // hipDoubleComplex *e;
    // hipMalloc((void**)&e, M*M*sizeof(hipDoubleComplex));
    // hipblasZgemm(
    //     cublasH,
    //     HIPBLAS_OP_N,HIPBLAS_OP_N,
    //     M,M,M,
    //     &alpha,
    //     AA,M,
    //     A_inv,M,
    //     &beta,
    //     e,M
    // );
    // hipMemcpy(test,e,M*M*sizeof(hipDoubleComplex),hipMemcpyDeviceToHost);
    // for(int i=0;i<32;i++){
    //     printf("E :%d: %f %f\n",i,test[i].x,test[i].y);
    // }
    



    VE_update<<<use*2, dim3(M,M)>>>(v_u, v_d, AA, A_inv, B, a, b, e_u, e_d, use);

    // hipDoubleComplex ff[1024], vd[4];
    // double eu[4],ed[4];
    // hipMemcpy(ff,v_u,M*use*sizeof(hipDoubleComplex),hipMemcpyDeviceToHost);
    // hipMemcpy(vd,v_d,use*sizeof(hipDoubleComplex),hipMemcpyDeviceToHost);
    // hipMemcpy(eu,e_u,use*sizeof(double),hipMemcpyDeviceToHost);
    // hipMemcpy(ed,e_d,use*sizeof(double),hipMemcpyDeviceToHost);

    // for(int i=0;i<4;i++){
    //     printf("v_u :%d: %f %f\n",i,ff[i].x,ff[i].y);
    // }

    // for(int i=0;i<4;i++){
    //     printf("e_u :%d: %f\n",i,eu[i]);
    // }

    hipFree(A);

}


//计算f

__global__ void f_cal( 
    double * f,   
    double * w_u,
    double * w_d,
    double * e_u,
    double * e_d,
    double F_norm,
    double p,
    int use,
    int use_norm){
    
    int m = threadIdx.x;
    int n = threadIdx.y;

    extern __shared__ double sh[];

    if(m == 0){
        sh[m*use+n] = w_u[n] * e_u[n];
    }
    else if (m == 1) {
        sh[m*use+n] = -1.0 * __log2f(w_u[n]);
    }
    else if (m == 2) {
        sh[m*use+n] = w_d[n] * e_d[n];
    }
    else if (m == 3) {
        sh[m*use+n] = -1.0 * __log2f(w_d[n]);
    }
    else
        sh[m*use+n] = 0.0;
    __syncthreads();

    int tid = m*use+n;
    for (int i = 2*use_norm; i > 0; i >>= 1)
    {
        if(tid < i){
            sh[tid] = sh[tid] + sh[tid+i];
        }
        __syncthreads();
        /* code */
    }
    __syncthreads();
    if(tid == 0)
        f[0] = sh[0] + (0.5 / p) * F_norm * F_norm; 
    
}

double F_cal(
    double * w_u,
    double * w_d,
    double * e_u,
    double * e_d,
    hipDoubleComplex * F,
    hipDoubleComplex * F_RF,
    hipDoubleComplex * F_BB,
    hipDoubleComplex * lambda,
    int use,
    double p,
    int RF,
    hipblasHandle_t cublasH,
    int use_norm){
    
    int M = 32;

    hipDoubleComplex * F_gap;
    double F_norm;
    hipMalloc((void**)&F_gap, M*use*sizeof(hipDoubleComplex));
    hipMemcpy(F_gap, F, M*use*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);
    hipDoubleComplex alpha = {-1.0,0.0};
    hipDoubleComplex beta = {1.0,0.0};
    hipDoubleComplex beta_p = {p,0.0};
    
    hipblasZgemm(
        cublasH,
        HIPBLAS_OP_N,HIPBLAS_OP_N,
        M,use,RF,
        &alpha,
        F_RF, M,
        F_BB, RF,
        &beta,
        F_gap,M
    );

    hipblasZgeam(
        cublasH,
        HIPBLAS_OP_N,HIPBLAS_OP_N,
        M,use,
        &beta,
        F_gap,M,
        &beta_p,
        lambda,M,
        F_gap,M
    );

    hipblasDznrm2(
        cublasH, M*use,
        F_gap, 1, &F_norm
    );
    double * f;
    hipMalloc((void **)&f,sizeof(double));
    f_cal<<<1,dim3(use_norm,use),use_norm*use>>>(f, w_u, w_d, e_u, e_d, F_norm, p, use,use_norm);

    double f_h;
    hipMemcpy(&f_h, f, sizeof(double), hipMemcpyDeviceToHost);
    return f_h;

}



//cv_cal

__global__ void norm_Inf(hipDoubleComplex * A,int M, int N,double * res ){
    int m = threadIdx.x; 
    int n = threadIdx.y;

    int len = M*N;
    int idx = m*16+n;
    __shared__ double A_s[512];
    if(idx < len){
        A_s[idx] = hipCabs(A[idx]);
    }
    else{
        A_s[idx] = 0.0f;
    }
    __syncthreads();

    for (int i = 256; i > 0; i >>= 1)
    {
        if(idx < i)
            A_s[idx] = max(A_s[idx],A_s[idx+i]);
        /* code */
        __syncthreads();
    }
    __syncthreads();

    res[0] = A_s[0];
        
}


double cv_cal(
    hipDoubleComplex * F,
    hipDoubleComplex * F_RF,
    hipDoubleComplex * F_BB,
    int M,
    int RF,
    int use,
    hipblasHandle_t cublasH,
    int use_norm){
    
    hipDoubleComplex * A;
    hipMalloc((void**)&A,M*use*sizeof(hipDoubleComplex));
    hipMemcpy(A,F,M*use*sizeof(hipDoubleComplex),hipMemcpyHostToHost);

    hipDoubleComplex alpha = {-1.0,0.0};
    hipDoubleComplex beta = {1.0,0.0};

    hipblasZgemm(
        cublasH,
        HIPBLAS_OP_N,HIPBLAS_OP_N,
        M,use,RF,
        &alpha,
        F_RF, M,
        F_BB, RF,
        &beta,
        A,M
    );

    double * inf_norm;
    hipMalloc((void**)&inf_norm,sizeof(double));

    norm_Inf<<<1,dim3(M,16)>>>(A, M, use, inf_norm);

    double cv;
    hipMemcpy(&cv,inf_norm,sizeof(double),hipMemcpyDeviceToHost);
    return cv;

}


void lambda_update(
    hipDoubleComplex * F,
    hipDoubleComplex * F_RF,
    hipDoubleComplex * F_BB,
    hipDoubleComplex * lambda,
    double p,
    int M,
    int RF,
    int use,
    hipblasHandle_t cublasH){

    hipDoubleComplex * A;
    hipMalloc((void**)&A,M*use*sizeof(hipDoubleComplex));
    hipMemcpy(A,F,M*use*sizeof(hipDoubleComplex),hipMemcpyHostToHost);

    hipDoubleComplex alpha = {-1.0,0.0};
    hipDoubleComplex beta = {1.0,0.0};
    hipDoubleComplex beta_p = {1/p,0.0};
    hipblasZgemm(
        cublasH,
        HIPBLAS_OP_N,HIPBLAS_OP_N,
        M,use,RF,
        &alpha,
        F_RF, M,
        F_BB, RF,
        &beta,
        A,M
    );
    hipblasZgeam(
        cublasH,
        HIPBLAS_OP_N,HIPBLAS_OP_N,
        M,use,
        &beta,
        lambda,M,
        &beta_p,
        A,M,
        lambda,M
    );

    // hipDoubleComplex test[1024];
    // hipMemcpy(test,lambda,M*use*sizeof(hipDoubleComplex),hipMemcpyDeviceToHost);
    // for(int i=0;i<32;i++){
    //     printf("lambda :%d: %e %e\n",i,test[i].x,test[i].y);
    // }
}
